#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#define BLOCK_SIZE 16;
const int N = 5;

//print the matrix.
template<class T>
void print(T A[N][N]) {
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
            std::cout << A[i][j] << " ";
        std::cout << std::endl;
    }
    std::cout << std::endl;
}


__global__ void child(int(*c)[N], const int(*a)[N], const int(*b)[N], int i) {
    int j = threadIdx.x;
    if (j < N)
        c[i][j] = a[i][j] + b[i][j];
}

__global__ void parent(int (* c)[N], const int (* a)[N], const int (* b)[N])
{
    int i = threadIdx.x;
    if(i<N)
    child <<<1,BLOCK_SIZE>>> (c, a, b, i);
}

int main()
{
    const int a[N][N] = { {1,2,3,4,5},{6,7,8,9,10},{11,12,13,14,15},{16,17,18,19,20},{21,22,23,24,25} };
    const int b[N][N] = { {1,2,3,4,5},{6,7,8,9,10},{11,12,13,14,15},{16,17,18,19,20},{21,22,23,24,25} };
    int c[N][N] = { {0},{0},{0},{0},{0} };

    // Cuda pointerek
    int(*d_a)[N], (*d_b)[N], (*d_c)[N];
    size_t bytes = N * N * sizeof(int);
    hipMalloc((void**)d_a, bytes);
    hipMalloc((void**)d_b, bytes);
    hipMalloc((void**)d_c, bytes);

    // Adatok másolása
    hipMemcpy(d_a, a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, bytes, hipMemcpyHostToDevice);


    // Kernel hívás
    parent <<<1, BLOCK_SIZE>>>(d_a,d_b,d_c);

    // Feldolgozott adat átvitele a GPU-ról
    hipMemcpy(c, d_c, bytes, hipMemcpyDeviceToHost);

    print(c);

    // Ideiglenes adattárolók felszabadítása
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_b);


    return 0;
}

